#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctype.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <unistd.h>
#include <fstream>

#define MAX_BOUNDARY    100
#define INIT_TEMP       20

/* 
  function: checkArgs
  ensures the command line argument fits requirements, 
  saves unsigned in placeholder and aborts otherwise
*/
void checkArgs(const char * arg, unsigned & placeholder) 
{
    /* check that all chars in arg are numbers */
    for (int i = 0; i < strlen(arg); i++) 
    {
        if (int(arg[i]) < 48 || int(arg[i]) > 57) 
        {
            std::cout << "Invalid parameters, please check your values." << std::endl;
            abort();
        }
    }

    /* convert to unsigned long */
    placeholder = strtoul(arg, NULL, 10);

    /* check that unsigned long is not zero or negative */
    if (placeholder <= 0) 
    {
        std::cout << "Invalid parameters, please check your values." << std::endl;
        abort();
    }
}

/* 
  function: calculateTemp
  
*/
__global__ void calculateTemp(double * h, unsigned N, unsigned I)
{
    while (I > 0)
    {
        double * g = (double *) malloc(N * N * sizeof(double));

        for (int i = 1; i < (N - 1); i++)
        {
            for (int j = 1; j < (N - 1); j++)
            {
                *(g + (j * N) + i) = 0.25 * *(h + (j * N) + i - 1) + *(h + (j * N) + i + 1) + *(h + ((j - 1) * N) + i) + *(h + ((j + 1) * N) + i);
            }
        }

        for (int i = 1; i < (N - 1); i++)
        {
            for (int j = 1; j < (N - 1); j++)
            {
                *(h + (j * N) + i) = *(g + (j * N) + i);
            }
        }

        free(g);
        I--;
    }
}

/* function: main
   
*/
int main(int argc, char * argv[])
{
    /* check arguments */
    unsigned N;
    unsigned I;
    int c;
    while ((c = getopt(argc, argv, "NI:")) != -1)
    {
        switch (c)
        {
            case 'N':
                checkArgs(optarg, N);
                N = N + 2;
                break;
            case 'I':
                checkArgs(optarg, I);
                break;
            case '?':
                std::cout << "Invalid parameters, please check your values." << std::endl;
                return 1;
            default:
                break;
        }
    }

    /* allocate cpu mem to array */
    size_t size = N * N * sizeof(double);
    double * H_h = (double *) malloc(size);

    /* initialize array interior values */
    for (int i = 1; i < (N - 1); i++)
    {
        for (int j = 1; j < (N - 1); j++)
        {
            *(H_h + (j * N) + i) = INIT_TEMP;
        }
    }

    /* initialize array exterior values */
    for (int i = 0; i < N; i++)
    {
        if (i > (0.3 * (N - 1)) && i < (0.7 * (N - 1)))
        {
            *(H_h + i) = MAX_BOUNDARY;
        }
        else
        {
            *(H_h + i) = INIT_TEMP;
        }
        *(H_h + ((N - 1) * N) + i) = INIT_TEMP;
    }
    for (int j = 1; j < (N - 1); j++)
    {
        *(H_h + (j * N)) = INIT_TEMP;
        *(H_h + (j * N) + N - 1) = INIT_TEMP;
    }

    /* get device properties to set num of threads and blocks */
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    int maxThreadsPerBlock = properties.maxThreadsPerBlock;

    /* allocate gpu mem to array */
    double * D_h = NULL;
    hipMalloc((void **) &D_h, size);

    /* send cpu data to gpu */
    hipMemcpy(D_h, H_h, size, hipMemcpyHostToDevice);

    /* initialize event timing variables */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* execute kernel with timing */
    hipEventRecord(start);
    calculateTemp<<<1, maxThreadsPerBlock>>>(D_h, N, I);
    hipEventRecord(stop);

    /* synchronize gpu & event at end of kernel execution */
    hipDeviceSynchronize();
    hipEventSynchronize(stop);

    /* calculate kernel execution time & write to console */
    float milli = 0;
    hipEventElapsedTime(&milli, start, stop);
    std::cout << std::fixed << std::setprecision(2) << milli;

    /* copy data back to cpu */
    hipMemcpy(H_h, D_h, size, hipMemcpyDeviceToHost);

    /* evaluate results & write to csv */
    std::ofstream ofs ("finalTemperatures.csv", std::ofstream::trunc);
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            ofs << std::fixed << std::setprecision(8) << *(H_h + (j * N) + i) << ',';
        }
        ofs << std::endl;
    }
    ofs.close();

    /* free allocated memory */
    hipFree(D_h);
    free(H_h);
    
    /* return successfully */
    return 0;
}