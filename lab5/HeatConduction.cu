#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctype.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <unistd.h>
#include <fstream>
#include <hip/hip_runtime.h>

#define MAX_BOUNDARY    100
#define INIT_TEMP       20

/* 
  function: checkArgs
  ensures the command line argument fits requirements
*/
unsigned checkArgs(const char * arg) 
{
    /* check that all chars in arg are numbers */
    for (int i = 0; i < strlen(arg); i++) 
    {
        if (int(arg[i]) < 48 || int(arg[i]) > 57) 
        {
            std::cout << "Invalid parameters, please check your values." << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    /* convert to unsigned long */
    unsigned val = strtoul(arg, NULL, 10);

    /* check that unsigned long is not zero or negative */
    if (val <= 0) 
    {
        std::cout << "Invalid parameters, please check your values." << std::endl;
        exit(EXIT_FAILURE);
    }

    return val;
}

/* 
  function: calculateTemp
  
*/
__global__ void calculateTemp(double * h, double * g, unsigned N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if ((i % (N - 1)) != 0 && (i % N) != 0 && i > N && i < (N * (N - 1)))
    {
        g[i] = 0.25 * (h[i- 1] + h[i + 1] + h[i - N] + h[i + N]);
    }
}

/* 
  function: copyMatrix
  
*/
__global__ void copyMatrix(double * h, double * g, unsigned N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    h[i] = g[i];
}

/* function: main
   
*/
int main(int argc, char * argv[])
{
    /* check arguments */
    unsigned N = 100;
    unsigned I = 1;

    // for(int i = 1; i < argc; i++)
    // {

    // }
    
    /* placeholder for error checking */
    hipError_t err = hipSuccess;

    /* allocate cpu mem to array */
    size_t size = N * N * sizeof(double);
    int numElements = (N - 2) * (N - 2);
    double * H_h = (double *) malloc(size);
    double * H_g = (double *) malloc(size);

    /* verify that allocation succeeded */
    if (H_h == NULL || H_g == NULL) 
    {
        std::cout << "Failed to allocate host matrices" << std::endl;
        exit(EXIT_FAILURE);
    }

    /* initialize all array interior positions */
    for (int i = 0; i < (N * N - 1); i++)
    {
        if (i > (0.3 * (N - 1)) && i < (0.7 * (N - 1)))
        {
            H_h[i] = MAX_BOUNDARY;
            H_g[i] = MAX_BOUNDARY;
        }
        else
        {
            H_h[i] = INIT_TEMP;
            H_g[i] = INIT_TEMP;
        }
    }

    /* set small section to 100 celsius */
    for (int i = ((int) ); i < ((int) ); i++)
    {
        
    }

    /* allocate gpu mem to matrix H */
    double * D_h = NULL;
    err = hipMalloc((void **) &D_h, size);

    // /* verify that allocation succeeded */
    // if (err != hipSuccess) 
    // {
    //     std::cout << "Failed to allocate device matrix H - " << hipGetErrorString(err) << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    /* allocate gpu mem to matrix G */
    double * D_g = NULL;
    err = hipMalloc((void **) &D_g, size);

    // /* verify that allocation succeeded */
    // if (err != hipSuccess) 
    // {
    //     std::cout << "Failed to allocate device matrix G - " << hipGetErrorString(err) << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    /* send matrix H to gpu */
    err = hipMemcpy(D_h, H_h, size, hipMemcpyHostToDevice);

    // /* verify memcopy from host to device succeeded */
    // if (err != hipSuccess) 
    // {
    //     std::cout << "Failed to copy matrix H from host to device - " << hipGetErrorString(err) << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    /* send matrix G to gpu */
    err = hipMemcpy(D_g, H_g, size, hipMemcpyHostToDevice);

    // /* verify memcopy from host to device succeeded */
    // if (err != hipSuccess) 
    // {
    //     std::cout << "Failed to copy matrix G from host to device - " << hipGetErrorString(err) << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    /* initialize event timing variables */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* get device properties to set num of threads and blocks */
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    int threadsPerBlock = properties.maxThreadsPerBlock;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    /* execute kernel with in loop with timing */
    hipEventRecord(start);
    while(I > 0)
    {
        calculateTemp<<<blocksPerGrid, threadsPerBlock>>>(D_h, D_g, N);
        hipDeviceSynchronize();
        copyMatrix<<<blocksPerGrid, threadsPerBlock>>>(D_h, D_g, N);
        hipDeviceSynchronize();
        I--;
    }
    hipEventRecord(stop);

    /* synchronize event at end of kernel execution */
    hipEventSynchronize(stop);

    // /* verify successful launch */
    // err = hipGetLastError();

    // if (err != hipSuccess) 
    // {
    //     std::cout << "Failed to launch calculateTemp kernel - " << hipGetErrorString(err) << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    /* calculate kernel execution time & write to console */
    float milli = 0;
    hipEventElapsedTime(&milli, start, stop);
    std::cout << std::fixed << std::setprecision(2) << milli << std::endl;

    /* copy data back to cpu */
    err = hipMemcpy(H_h, D_h, size, hipMemcpyDeviceToHost);

    // /* verify memcopy from device to host succeeded */
    // if (err != hipSuccess) 
    // {
    //     std::cout << "Failed to copy matrix H from device to host - " << hipGetErrorString(err) << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    /* evaluate results & write to csv */
    std::ofstream ofs ("finalTemperatures.csv", std::ofstream::trunc);
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            ofs << std::fixed << std::setprecision(6) << *(H_h + (j * N) + i) << ", ";
        }
        ofs << std::endl;
    }
    ofs.close();

    /* free device memory */
    err = hipFree(D_h);
    
    // /* verify cuda memory successfully freed */
    // if (err != hipSuccess) 
    // {
    //     std::cout << "Failed to free device matrix/matrices - " << hipGetErrorString(err) << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    /* free device memory */
    err = hipFree(D_g);
    
    // /* verify cuda memory successfully freed */
    // if (err != hipSuccess) 
    // {
    //     std::cout << "Failed to free device matrix/matrices - " << hipGetErrorString(err) << std::endl;
    //     exit(EXIT_FAILURE);
    // }
    
    /* free host memory */
    free(H_h);
    free(H_g);

    /* return successfully */
    exit(EXIT_SUCCESS);
}