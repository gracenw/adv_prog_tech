#include <stdio.h>
#include <ctype.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <unistd.h>
#include <fstream>
#include <hip/hip_runtime.h>

#define MAX_BOUNDARY    100
#define INIT_TEMP       20

/* 
  function: checkArgs
  ensures the command line argument fits requirements
*/
unsigned checkArgs(const char * arg) 
{
    /* check that all chars in arg are numbers */
    for (int i = 0; i < strlen(arg); i++) 
    {
        if (int(arg[i]) < 48 || int(arg[i]) > 57) 
        {
            std::cout << "Invalid parameters, please check your values." << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    /* convert to unsigned long */
    unsigned val = strtoul(arg, NULL, 10);

    /* check that unsigned long is not zero or negative */
    if (val <= 0) 
    {
        std::cout << "Invalid parameters, please check your values." << std::endl;
        exit(EXIT_FAILURE);
    }

    return val;
}

/* 
  function: calculateTemp
  
*/
__global__ void calculateTemp(double * h, double * g, unsigned N)
{
    int i = blockIdx.x;

    if (((i + 1) % N) != 0 && (i % N) != 0 && i > N && i < (N * (N - 1)))
    {
        g[i] = 0.25 * (h[i- 1] + h[i + 1] + h[i - N] + h[i + N]);
    }
}

/* 
  function: copyMatrix
  
*/
__global__ void copyMatrix(double * h, double * g, unsigned N)
{
    int i = blockIdx.x;

    h[i] = g[i];
}

/* function: main
   
*/
int main(int argc, char * argv[])
{
    /* check arguments */
    unsigned N = 0;
    unsigned I = 0;

    for (int i = 1; i < argc; i += 2)
    {
        if (strcmp(argv[i], "-N") == 0)
        {
            N = checkArgs(argv[i + 1]);
            N = N + 2;
        }
        else if (strcmp(argv[i], "-I") == 0)
        {
            I = checkArgs(argv[i + 1]);
        }
        else
        {
            std::cout << "Invalid parameters, please check your values." << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    /* allocate cpu mem to array */
    size_t size = N * N * sizeof(double);
    double * H_h = (double *) malloc(size);
    double * H_g = (double *) malloc(size);

    /* initialize all array interior positions */
    for (int i = 0; i < (N * N); i++)
    {
        if (i > (0.3 * (N - 1)) && i < (0.7 * (N - 1)))
        {
            H_h[i] = MAX_BOUNDARY;
            H_g[i] = MAX_BOUNDARY;
        }
        else
        {
            H_h[i] = INIT_TEMP;
            H_g[i] = INIT_TEMP;
        }
    }

    /* allocate gpu mem to matrix H */
    double * D_h = NULL;
    hipMalloc((void **) &D_h, size);

    /* allocate gpu mem to matrix G */
    double * D_g = NULL;
    hipMalloc((void **) &D_g, size);

    /* send matrix H to gpu */
    hipMemcpy(D_h, H_h, size, hipMemcpyHostToDevice);

    /* send matrix G to gpu */
    hipMemcpy(D_g, H_g, size, hipMemcpyHostToDevice);

    /* initialize event timing variables */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* get device properties to set num of threads and blocks */
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    int threadsPerBlock = properties.maxThreadsPerBlock;

    /* execute kernel with in loop with timing */
    hipEventRecord(start);
    while(I > 0)
    {
        calculateTemp<<<(N * N), threadsPerBlock>>>(D_h, D_g, N);
        hipDeviceSynchronize();
        copyMatrix<<<(N * N), threadsPerBlock>>>(D_h, D_g, N);
        hipDeviceSynchronize();
        I--;
    }
    hipEventRecord(stop);

    /* synchronize event at end of kernel execution */
    hipEventSynchronize(stop);

    /* calculate kernel execution time & write to console */
    float milli = 0;
    hipEventElapsedTime(&milli, start, stop);
    std::cout << std::fixed << std::setprecision(2) << milli << std::endl;

    /* copy data back to cpu */
    hipMemcpy(H_h, D_h, size, hipMemcpyDeviceToHost);

    /* evaluate results & write to csv */
    std::ofstream ofs ("finalTemperatures.csv", std::ofstream::trunc);
    for (int i = 0; i < (N * N); i++)
    {
        ofs << std::fixed << std::setprecision(6) << H_h[i];
        if (((i + 1) % N) == 0 && i != 0)
        {
            ofs << std::endl;
        }
        else
        {
            ofs << ", ";
        }
    }
    ofs.close();

    /* free device memory */
    hipFree(D_h);
    hipFree(D_g);
    
    /* free host memory */
    free(H_h);
    free(H_g);

    /* return successfully */
    exit(EXIT_SUCCESS);
}